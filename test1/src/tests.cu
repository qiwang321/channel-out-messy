#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <stdio.h>
#include <vector>
#include <time.h>

#include <nvmatrix.cuh>
#include <cudaconv2.cuh>
#include "routines.cuh"
#include <layer_kernels.cuh>

using namespace std;

extern LayerOpt opt1, opt2, opt3, opt4, optTop;
extern FILE* pFile;
//void assignOpt();

// test blocks
void testSamplePatches() {
	printf("testing samplePatches()...\n");
	int patchSize = 16;
	int numPatches = 10000;
	int dims[3] = {32, 32, 3};
	int numRecords = 1000;
	char* in_name = "/scratch0/qwang37/cifar-10-batches-bin/cifar_normalized.bin";
	char* out_name1 = "/scratch0/qwang37/cifar-10-batches-bin/cifar_samplePatch_test_half.bin";
	samplePatches(patchSize, numPatches, dims, numRecords, in_name, out_name1);
	patchSize = 32;
	char* out_name2 = "/scratch0/qwang37/cifar-10-batches-bin/cifar_samplePatch_test_full.bin";
	samplePatches(patchSize, numPatches, dims, numRecords, in_name, out_name2);
	printf("samplePatches() test done!\n");
}

void testMult() {
	printf("rightMult test\n");
	MTYPE data1[6] = {0,1,2,3,4,5};
	Matrix m1(data1, 2, 3, false);
	NVMatrix nvm1(m1, true);

	MTYPE data2[9] = {0,1,2,3,4,5,6,7,8};
	Matrix m2(data2, 3, 3, false);
	NVMatrix nvm2(m2, true);

	NVMatrix nvm3;
	nvm1.rightMult(nvm2,1);

	nvm1.printShape("nvm1");
	printf("nvm1 _isTrans = %d\n", nvm1.isTrans());
	nvm1.print(nvm1.getNumRows(), nvm1.getNumCols());

}

void testConv() {
	printf("convolution test\n");
	MTYPE* data = (MTYPE*) malloc(8*3*3*2*sizeof(MTYPE));
	for (int i = 1; i <= 8*3*3*2; i++) {
		data[i-1] = i; // first image is a bunch of "0"s; second image is a bunch of "1"s
	}
	Matrix *im = new Matrix(data, 8*3*3, 2, true); // the transpose specification does not have effect here!
	NVMatrix *nvim = new NVMatrix(*im, true);

	printf("filters\n");
	MTYPE* data2 = (MTYPE*) malloc(32*4*2*sizeof(MTYPE));
	for (int i = 1; i <= 32*4*2; i++) {
		data2[i-1] = i; // filters are a bunch of "1"s
	}
	Matrix *f = new Matrix(data2, 32*4, 2, true);
	NVMatrix *nvf = new NVMatrix(*f, true);

	NVMatrix *targets = new NVMatrix();

	convWeightActs(*nvim, *nvf, *targets, 3, 2, 2, 2, 0, 1, 8, 2, 0);
	printf("numRows: %d, numCols: %d\n", targets->getNumRows(), targets->getNumCols());
	targets->print(targets->getNumRows(), targets->getNumCols());

}

void testMatrixIO() {
	printf("testing Matrix IO...\n");
	MTYPE data[8] = {0, 1, 2, 3, 4, 5, 6, 7};
	Matrix rm(data, 2, 4, true);
	Matrix cm(data, 4, 2, false);
	NVMatrix NVrm(rm, true);
	NVMatrix NVcm(cm, true);
	NVSaveToFile(NVrm, "NVrm.bin");
	NVSaveToFile(NVcm, "NVcm.bin");
	/* attention: the Matrix and NVMatrix classes do not
	have proper direct copying assignment operators!
	assignment has to be done through reference
	 */
	NVMatrix NVrm1(2,4), NVcm1(4,2);
	NVrm1.setTrans(true); NVcm1.setTrans(false);
	NVReadFromFile(NVrm1, "NVrm.bin");
	NVReadFromFile(NVcm1, "NVcm.bin");
	NVrm1.printShape("NVrm1");
	NVrm1.print(NVrm1.getNumRows(), NVrm1.getNumCols());
	NVcm1.printShape("NVcm1");
	NVcm1.print(NVcm1.getNumRows(), NVcm1.getNumCols());
	printf("Matrix IO test complete!\n");
}

void testDataIO() {
	printf("testing Data IO...\n");
	MTYPE data[8] = {0, 1, 2, 3, 4, 5, 6, 7};
	Matrix rm(data, 2, 4, true);
	Matrix cm(data, 4, 2, false);
	NVMatrix NVrm(rm, true);
	NVMatrix NVcm(cm, true);
	NVSaveToFile(NVrm, "NVm.bin", true);
	NVSaveToFile(NVcm, "NVm.bin", true);
	/* attention: the Matrix and NVMatrix classes do not
	have proper direct copying assignment operators!
	assignment has to be done through reference
	 */
	NVMatrix NVrm1(2,4), NVcm1(4,2);
	NVrm1.setTrans(true); NVcm1.setTrans(false);
	NVReadFromFile(NVrm1, "NVm.bin");
	NVReadFromFile(NVcm1, "NVm.bin", 2);
	NVrm1.printShape("NVrm1");
	NVrm1.print(NVrm1.getNumRows(), NVrm1.getNumCols());
	NVcm1.printShape("NVcm1");
	NVcm1.print(NVcm1.getNumRows(), NVcm1.getNumCols());
	printf("Data IO test complete!\n");
}

/*
void testTrainFCAE() {
	printf("testing trainFCAE()...\n");
	LayerOpt opt1;
	char* layerName = "layer1";
	char* df = "/scratch0/qwang37/cifar-10-batches-bin/cifar_patches.bin";
	opt1.layerName = layerName;
	opt1.dataFile = df;
	opt1.patchSize = 5;
	opt1.numChannels = 3;
	opt1.numFilters = 64;
	opt1.batchSize = 2000;
	opt1.batchNum = 1000;
	opt1.numEpochs = 100;
	opt1.initstv = 0.01;
	opt1.mom = 0.0;
	opt1.lrW = 0.01;
	opt1.lrB = 0.01;
	opt1.weightDecay = 0.003;
	opt1.sparseParam = 0.035;
	opt1.sparseWeight = 0.0;
	NVMatrix weight1; NVMatrix bias1; // parameters for the first layer
	char* weightFile ="/scratch0/qwang37/cifar-10-batches-bin/filters5x5_layer1.bin";
	char* biasFile = "/scratch0/qwang37/cifar-10-batches-bin/biases5x5_layer1.bin";
	trainFCAE(opt1, weight1, bias1, weightFile, biasFile);
	//weight1.printShape("forward weight");
	//weight1.print(weight1.getNumRows(), weight1.getNumCols());
	printf("trainFCAE() test complete!\n");
}
 */

/*
void testGenerateDataConv(char* poolType) {
	printf("testing testGenerateDataConv(%s)...\n", poolType);
	Dim dims;
	dims.dataX = 32; dims.dataY = 32; dims.dataC = 3; dims.batchSize = 5000; dims.numBatches = 10;
	dims.filterX = 5; dims.numFilters = 64; dims.stride = 1; dims.padding = 0;
	dims.poolSize = 3; dims.poolStride = 2; dims.poolStartX = 0; strcpy(dims.pooler, poolType);
	dims.poolOutX = (dims.dataX - dims.filterX + 1 - dims.poolSize) / dims.poolStride + 1;

	char* sourceFile = "/scratch0/qwang37/cifar-10-batches-bin/cifar_normalized.bin";
	char* destFile = (char*) malloc (100);
	strcpy(destFile, "/scratch0/qwang37/cifar-10-batches-bin/cifar_layer2data_conv_");
	strcat(destFile, poolType);
	strcat(destFile, ".bin");
	remove(destFile);

	char* layerType = "conv";
	NVMatrix weight(dims.filterX*dims.filterX*dims.dataC, dims.numFilters), biases(1, dims.numFilters);
	NVReadFromFile(weight, "/scratch0/qwang37/cifar-10-batches-bin/filters5x5_layer1.bin");
	NVReadFromFile(biases, "/scratch0/qwang37/cifar-10-batches-bin/biases5x5_layer1.bin");
	generateData(sourceFile, destFile, layerType, weight, biases, dims);
	printf("testGenerateDataConv() test complete!\n");
}
 */


/*
void testGenerateDataFC() {
	printf("testing testGenerateDataFC()...\n");
	Dim dims;
	dims.dataX = 5; dims.dataY = 5; dims.dataC = 3; dims.batchSize = 100000; dims.numBatches = 5;
	dims.filterX = 5; dims.numFilters = 64; dims.stride = 1; dims.padding = 0;

	char* sourceFile = "/scratch0/qwang37/cifar-10-batches-bin/cifar_patches.bin";
	char* destFile = "/scratch0/qwang37/cifar-10-batches-bin/cifar_layer2data_FC.bin";
	char* layerType = "FC";
	NVMatrix weight(dims.filterX*dims.filterX*dims.dataC, dims.numFilters), biases(1, dims.numFilters);
	NVReadFromFile(weight, "/scratch0/qwang37/cifar-10-batches-bin/filters5x5_layer1.bin");
	NVReadFromFile(biases, "/scratch0/qwang37/cifar-10-batches-bin/biases5x5_layer1.bin");
	remove(destFile);
	generateData(sourceFile, destFile, layerType, weight, biases, dims);
	printf("testGenerateDataFC() test complete!\n");
}
 */

void testNVLabelReadFromFile() {
	printf("testing NVLabelReadFromFile()...\n");
	NVMatrix labels(10,10);
	labels.setTrans(false);
	NVLabelReadFromFile(labels, "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", 101);
	labels.printShape("labels");
	labels.print(10,10);
	printf("NVLabelReadFromFile() test complete!\n");
}

void testNVRawLabelReadFromFile() {
	printf("testing NVRawLabelReadFromFile()...\n");
	NVMatrix labels(1,10);
	NVRawLabelReadFromFile(labels, "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", 101);
	labels.printShape("labels");
	labels.print(1,10);
	printf("NVRawLabelReadFromFile() test complete!\n");
}



void finetune_rnorm() {
	////assignOpt();
	printf("starting finetune_rnorm()!\n");
	fprintf(pFile, "starting finetune_rnorm!\n");

	// initialize cublas
	hipSetDevice(cutGetMaxGflopsDeviceId());
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	cublasInit();

	// data and parameters storage
	NVMatrix act1, act1Pool, act1PoolNorm, act1Denom;
	NVMatrix act2, act2Norm, act2NormPool, act2Denom;
	NVMatrix act3;
	NVMatrix act4;
	NVMatrix actTop;
	NVMatrix act1Grad, act1PoolGrad, act1PoolNormGrad;
	NVMatrix act2Grad, act2NormGrad, act2NormPoolGrad;
	NVMatrix act3Grad;
	NVMatrix act4Grad;
	NVMatrix actTopGrad;

	NVMatrix weight1, weight2, weight3, weight4, weightTop;
	NVMatrix weight1Grad, weight2Grad, weight3Grad, weight4Grad, weightTopGrad;
	NVMatrix weight1Inc, weight2Inc, weight3Inc, weight4Inc, weightTopInc;
	NVMatrix weight1GradTmp, weight2GradTmp, weight3GradTmp, weight4GradTmp, weightTopGradTmp;

	NVMatrix bias1, bias2, bias3, bias4, biasTop; // bias4 is just an all-zero dummy vector
	NVMatrix bias1Grad, bias2Grad, bias3Grad, bias4Grad, biasTopGrad;
	NVMatrix bias1Inc, bias2Inc, bias3Inc, bias4Inc, biasTopInc;

	// initialize parameters
	if (opt1.loadParam) {
		weight1.resize(opt1.numVis, opt1.numFilters);
		weight2.resize(opt2.numVis, opt2.numFilters);
		weight3.resize(opt3.numVis * opt3.outX * opt3.outX, opt3.numFilters);
		weight4.resize(opt4.numVis * opt4.outX * opt4.outX, opt4.numFilters);
		weightTop.resize(optTop.numVis, optTop.numFilters);

		bias1.resize(opt1.numFilters, 1);
		bias2.resize(opt2.numFilters, 1);
		bias3.resize(opt3.numFilters * opt3.outX * opt3.outX, 1);
		bias4.resize(opt4.numFilters * opt4.outX * opt4.outX, 1);
		biasTop.resize(1, optTop.numFilters);
		biasTop.setTrans(true);


		NVReadFromFile(weight1, "/scratch0/qwang37/cifar-10-batches-bin/weight1.bin");
		NVReadFromFile(weight2, "/scratch0/qwang37/cifar-10-batches-bin/weight2.bin");
		NVReadFromFile(weight3, "/scratch0/qwang37/cifar-10-batches-bin/weight3.bin");
		NVReadFromFile(weight4, "/scratch0/qwang37/cifar-10-batches-bin/weight4.bin");
		NVReadFromFile(weightTop, "/scratch0/qwang37/cifar-10-batches-bin/weightTop.bin");

		NVReadFromFile(bias1, "/scratch0/qwang37/cifar-10-batches-bin/bias1.bin");
		NVReadFromFile(bias2, "/scratch0/qwang37/cifar-10-batches-bin/bias2.bin");
		NVReadFromFile(bias3, "/scratch0/qwang37/cifar-10-batches-bin/bias3.bin");
		NVReadFromFile(bias4, "/scratch0/qwang37/cifar-10-batches-bin/bias4.bin");
		NVReadFromFile(biasTop, "/scratch0/qwang37/cifar-10-batches-bin/biasTop.bin");
	}
	else {
		initWeights(weight1, opt1.numVis, opt1.numFilters, false, opt1.initstv);
		initWeights(weight2, opt2.numVis, opt2.numFilters, false, opt2.initstv);
		initWeights(weight3, opt3.numVis * opt3.outX * opt3.outX, opt3.numFilters, false, opt3.initstv);
		initWeights(weight4, opt4.numVis * opt4.outX * opt4.outX, opt4.numFilters, false, opt4.initstv);
		initWeights(weightTop, optTop.numVis, optTop.numFilters, false, optTop.initstv);

		initWeights(bias1, opt1.numFilters, 1, false, 0.0);
		initWeights(bias2, opt2.numFilters, 1, false, 0.0);
		initWeights(bias3, opt3.numFilters * opt3.outX * opt3.outX, 1, false, 0.0);
		initWeights(bias4, opt4.numFilters * opt4.outX * opt4.outX, 1, false, 0.0);
		initWeights(biasTop, 1, optTop.numFilters, true, 0.0);
	}

	initWeights(weight1Inc, opt1.numVis, opt1.numFilters, false, 0.0); initWeights(weight1Grad, opt1.numVis, opt1.numFilters, false, 0.0);
	initWeights(weight2Inc, opt2.numVis, opt2.numFilters, false, 0.0); initWeights(weight2Grad, opt2.numVis, opt2.numFilters, false, 0.0);
	initWeights(weight3Inc, opt3.numVis * opt3.outX * opt3.outX, opt3.numFilters, false, 0.0); initWeights(weight3Grad, opt3.numVis * opt3.outX * opt3.outX, opt3.numFilters, false, 0.0); // not useful for 3 and 4
	initWeights(weight4Inc, opt4.numVis * opt4.outX * opt4.outX, opt4.numFilters, false, 0.0); initWeights(weight4Grad, opt4.numVis * opt4.outX * opt4.outX, opt4.numFilters, false, 0.0);
	initWeights(weightTopInc, optTop.numVis, optTop.numFilters, false, 0.0); initWeights(weightTopGrad, optTop.numVis, optTop.numFilters, false, 0.0);

	initWeights(bias1Inc, opt1.numFilters, 1, false, 0.0); initWeights(bias1Grad, opt1.numFilters, 1, false, 0.0);
	initWeights(bias2Inc, opt2.numFilters, 1, false, 0.0); initWeights(bias2Grad, opt2.numFilters, 1, false, 0.0);
	initWeights(bias3Inc, opt3.numFilters * opt3.outX * opt3.outX, 1, false, 0.0); initWeights(bias3Grad, opt3.numFilters * opt3.outX * opt3.outX, 1, false, 0.0); // not useful for 3
	initWeights(bias4Inc, opt4.numFilters * opt4.outX * opt4.outX, 1, false, 0.0); initWeights(bias4Grad, opt4.numFilters * opt4.outX * opt4.outX, 1, false, 0.0);
	initWeights(biasTopInc, 1, opt1.labelSize, true, 0.0); initWeights(biasTopGrad, 1, opt1.labelSize, true, 0.0);


	// read data to host memory (and labels to the GPU memory)
	int imPixels = 32*32*opt1.numChannels;
	int batchSize = opt1.batchSize;
	int trainBatchNum = opt1.numTrain / batchSize;
	int testBatchNum = opt1.numTest / batchSize;
	vector<Matrix*> CPUTrain(trainBatchNum), CPUTest(testBatchNum);
	vector<NVMatrix*> GPUTrain(trainBatchNum), GPUTest(testBatchNum);
	vector<NVMatrix*> GPURawLabelTrain(trainBatchNum), GPURawLabelTest(testBatchNum);

	for (int batch = 0; batch < trainBatchNum; batch++) {
		CPUTrain[batch] = new Matrix(imPixels, batchSize);
		CPUTrain[batch]->setTrans(false);
		GPUTrain[batch] = new NVMatrix();
		hmReadFromFile(*CPUTrain[batch], opt1.dataPath + "/cifar_raw.bin", batch*batchSize);
		GPURawLabelTrain[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTrain[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTrain[batch], opt1.dataPath + "/cifar_labels.bin", batch*batchSize);
	}
	batchSize = opt1.numTrain % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTrain.push_back(new Matrix(imPixels, batchSize));
		CPUTrain.back()->setTrans(false);
		GPUTrain.push_back(new NVMatrix());
		hmReadFromFile(*CPUTrain.back(), opt1.dataPath + "/cifar_raw.bin", trainBatchNum*batchSize);
		GPURawLabelTrain.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTrain.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTrain.back(), opt1.dataPath + "/cifar_labels.bin", trainBatchNum*batchSize);
	}
	// test set
	batchSize = opt1.batchSize;
	for (int batch = 0; batch < testBatchNum; batch++) {
		CPUTest[batch] = new Matrix(imPixels, batchSize);
		CPUTest[batch]->setTrans(false);
		GPUTest[batch] = new NVMatrix();
		hmReadFromFile(*CPUTest[batch], opt1.dataPath + "/cifar_raw.bin", opt1.numTrain+batch*batchSize);
		GPURawLabelTest[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTest[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest[batch], opt1.dataPath + "/cifar_labels.bin", opt1.numTrain+batch*batchSize);
	}
	batchSize = opt1.numTest % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTest.push_back(new Matrix(imPixels, batchSize));
		CPUTest.back()->setTrans(false);
		GPUTest.push_back(new NVMatrix());
		hmReadFromFile(*CPUTest.back(), opt1.dataPath + "/cifar_raw.bin", opt1.numTrain+testBatchNum*batchSize);
		GPURawLabelTest.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTest.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest.back(), opt1.dataPath + "/cifar_labels.bin", opt1.numTrain+testBatchNum*batchSize);
	}

	NVMatrix trueLabelLogProbs;
	NVMatrix correctProbs;
	MTYPE cost; // as before, we trace the performance using the cost variable
	MTYPE cost1;
	NVMatrix absM;
	MTYPE weightAbs1, weightAbs2, weightAbs3, weightAbs4, weightAbsTop;
	MTYPE biasAbs1, biasAbs2, biasAbs3, biasAbs4, biasAbsTop;
	MTYPE weightGradAbs1, weightGradAbs2, weightGradAbs3, weightGradAbs4, weightGradAbsTop;
	MTYPE biasGradAbs1, biasGradAbs2, biasGradAbs3, biasGradAbs4, biasGradAbsTop;
	clock_t startClock;
	clock_t tick;

	float lr_scale = 1.0, mom_scale = 1.0;

	cropDataProvider(CPUTest, GPUTest, opt1, true, opt1.whitened); // test data is fixed

	for (int epoch = 0; epoch < opt1.numEpochs; epoch++) {
		cost = 0;
		cost1 = 0;
		cropDataProvider(CPUTrain, GPUTrain, opt1, false, opt1.whitened); // copy data to the GPU side
		hipDeviceSynchronize();
		startClock = clock();
		for (int batch = 0; batch < GPUTrain.size(); batch++) {
			batchSize = GPUTrain[batch]->getNumCols();
			// ====forward pass====
			// 0->1
			//cout << "0->1\n";
			//original
			activateConv(*GPUTrain[batch], act1, weight1, bias1, opt1);
			act1.apply(ReluOperator());
			act1Pool.transpose(false);
			convLocalPool(act1, act1Pool, opt1.numFilters, opt1.poolSize, opt1.poolStartX, opt1.poolStride, opt1.poolOutX, MaxPooler());
			convResponseNormCrossMap(act1Pool, act1Denom, act1PoolNorm, opt1.numFilters, opt1.sizeF, opt1.addScale/opt1.sizeF, opt1.powScale, false);

			// 1->2
			//cout << "1->2\n";
			//original
			activateConv(act1PoolNorm, act2, weight2, bias2, opt2);
			act2.apply(ReluOperator());
			convResponseNormCrossMap(act2, act2Denom, act2Norm, opt2.numFilters, opt2.sizeF, opt2.addScale/opt2.sizeF, opt2.powScale, false);
			act2NormPool.transpose(false);
			convLocalPool(act2Norm, act2NormPool, opt2.numFilters, opt2.poolSize, opt2.poolStartX, opt2.poolStride, opt2.poolOutX, MaxPooler());

			// 2->3
			//cout << "2->3\n";
			// original
			activateLocal(act2NormPool, act3, weight3, bias3, opt3);
			act3.apply(ReluOperator());

			// 3->4
			//cout << "3->4\n";
			// original
			activateLocal(act3, act4, weight4, bias4, opt4);
			act4.apply(ReluOperator());

			// 4->top
			//cout << "4->top\n";
			actTop.transpose(true);
			actTop.resize(batchSize, opt1.labelSize);
			activate(act4, actTop, weightTop, biasTop, 0, 1);

			//softmax layer
			NVMatrix& max = actTop.max(1);
			actTop.addVector(max, -1);
			actTop.apply(NVMatrixOps::Exp());
			NVMatrix& sum = actTop.sum(1);
			actTop.eltwiseDivideByVector(sum);
			delete &max;
			delete &sum;

			// compute cost
			computeLogregSoftmaxGrad(*GPURawLabelTrain[batch], actTop, actTopGrad, false, 1);
			actTop.transpose(false);
			computeLogregCost(*GPURawLabelTrain[batch], actTop, trueLabelLogProbs, correctProbs); //labelLogProbs:(1, numCases); correctProbs:(1, numCases)
			cost += correctProbs.sum() / batchSize;
			cost1 += trueLabelLogProbs.sum() / batchSize;


			// ====== back pass ======
			// top -> 4, 3, 2, 1
			//cout << "top -> 4, 3, 2, 1";
			// weight update
			NVMatrix& act4T = act4.getTranspose();
			weightTopGrad.addProduct(act4T, actTopGrad, 0, 1);
			biasTopGrad.addSum(actTopGrad, 0, 0, 1);
			delete &act4T;

			// bp
			actTopGrad.transpose(true);
			NVMatrix& weightTopT = weightTop.getTranspose();
			act4Grad.addProduct(actTopGrad, weightTopT, 0, 1);
			delete &weightTopT;

			// 4->3
			//cout << "4->3\n";
			act4Grad.transpose(false); // convert back to row-major
			act4.transpose(false);
			act4Grad.applyBinary(ReluGradientOperator(), act4);

			localWeightActs(act3, act4Grad, weight4Grad, opt4.imSize, opt4.outX, opt4.outX, opt4.patchSize, opt4.paddingStart, 1, opt4.numChannels, 1);
			bias4Grad.addSum(act4Grad, 1, 0, 1);
			localImgActs(act4Grad, weight4, act3Grad, opt4.imSize, opt4.imSize, opt4.outX, opt4.paddingStart, 1, opt4.numChannels, 1);

			// 3->2
			//cout << "3->2\n";
			// original part
			act3Grad.transpose(false); // convert back to row-major
			act3.transpose(false);
			act3Grad.applyBinary(ReluGradientOperator(), act3);
			localWeightActs(act2NormPool, act3Grad, weight3Grad, opt3.imSize, opt3.outX, opt3.outX, opt3.patchSize, opt3.paddingStart, 1, opt3.numChannels, 1);
			bias3Grad.addSum(act3Grad, 1, 0, 1);
			localImgActs(act3Grad, weight3, act2NormPoolGrad, opt3.imSize, opt3.imSize, opt3.outX, opt3.paddingStart, 1, opt3.numChannels, 1);

			// 2->1
			//cout << "2->1\n";
			// original part
			act2NormPoolGrad.transpose(false);
			act2NormPool.transpose(false);
			convLocalMaxUndo(act2Norm, act2NormPoolGrad, act2NormPool, act2NormGrad, opt2.poolSize, opt2.poolStartX, opt2.poolStride, opt2.poolOutX);
			convResponseNormCrossMapUndo(act2NormGrad, act2Denom, act2, act2Norm, act2Grad, opt2.numFilters, opt2.sizeF, opt2.addScale/opt2.sizeF, opt2.powScale, false, 0, 1);
			act2Grad.applyBinary(ReluGradientOperator(), act2);
			convWeightActs(act1PoolNorm, act2Grad, weight2GradTmp, opt2.imSize, opt2.outX, opt2.outX, opt2.patchSize, opt2.paddingStart, 1, opt2.numChannels, 1, opt2.partialSum);
			weight2GradTmp.reshape(opt2.outX * opt2.outX / opt2.partialSum, opt2.numChannels * opt2.patchSize * opt2.patchSize * opt2.numFilters);
			weight2Grad.addSum(weight2GradTmp, 0, 0, 1);
			weight2Grad.reshape(opt2.numChannels * opt2.patchSize * opt2.patchSize, opt2.numFilters);
			act2Grad.reshape(opt2.numFilters, opt2.outX * opt2.outX * batchSize);
			bias2Grad.addSum(act2Grad, 1, 0, 1);
			act2Grad.reshape(opt2.numFilters * opt2.outX * opt2.outX, batchSize);
			convImgActs(act2Grad, weight2, act1PoolNormGrad, opt2.imSize, opt2.imSize, opt2.outX, opt2.paddingStart, 1, opt2.numChannels, 1);

			// 1->0
			//cout << "1->0\n";
			// original part
			act1PoolNormGrad.transpose(false);
			act1PoolNorm.transpose(false);
			convResponseNormCrossMapUndo(act1PoolNormGrad, act1Denom, act1Pool, act1PoolNorm, act1PoolGrad, opt1.numFilters, opt1.sizeF, opt1.addScale/opt1.sizeF, opt1.powScale, false, 0, 1);
			convLocalMaxUndo(act1, act1PoolGrad, act1Pool, act1Grad, opt1.poolSize, opt1.poolStartX, opt1.poolStride, opt1.poolOutX);
			act1Grad.applyBinary(ReluGradientOperator(), act1);
			convWeightActs(*GPUTrain[batch], act1Grad, weight1GradTmp, opt1.imSize, opt1.outX, opt1.outX, opt1.patchSize, opt1.paddingStart, 1, opt1.numChannels, 1, opt1.partialSum);
			weight1GradTmp.reshape(opt1.outX * opt1.outX / opt1.partialSum, opt1.numChannels * opt1.patchSize * opt1.patchSize * opt1.numFilters);
			weight1Grad.addSum(weight1GradTmp, 0, 0, 1);
			weight1Grad.reshape(opt1.numChannels * opt1.patchSize * opt1.patchSize, opt1.numFilters);
			act1Grad.reshape(opt1.numFilters, opt1.outX * opt1.outX * batchSize);
			bias1Grad.addSum(act1Grad, 1, 0, 1);
			act1Grad.reshape(opt1.numFilters * opt1.outX * opt1.outX, batchSize);

			// update
			lr_scale = lrDecay(lr_scale, opt1.lrDecayType, opt1.lrDecayFactor, opt1.lrMinRate);
			mom_scale = momInc(mom_scale, opt1.momIncType, opt1.momIncFactor, opt1.momMaxRate);
			updateWeight(weight1Grad, weight1Inc, weight1, opt1, batchSize, lr_scale, mom_scale);
			updateWeight(weight2Grad, weight2Inc, weight2, opt2, batchSize, lr_scale, mom_scale);
			updateWeight(weight3Grad, weight3Inc, weight3, opt3, batchSize, lr_scale, mom_scale);
			updateWeight(weight4Grad, weight4Inc, weight4, opt4, batchSize, lr_scale, mom_scale);
			updateWeight(weightTopGrad, weightTopInc, weightTop, optTop, batchSize, lr_scale, mom_scale);
			updateBias(bias1Grad, bias1Inc, bias1, opt1, batchSize, lr_scale, mom_scale);
			updateBias(bias2Grad, bias2Inc, bias2, opt2, batchSize, lr_scale, mom_scale);
			updateBias(bias3Grad, bias3Inc, bias3, opt3, batchSize, lr_scale, mom_scale);
			updateBias(bias4Grad, bias4Inc, bias4, opt4, batchSize, lr_scale, mom_scale);
			updateBias(biasTopGrad, biasTopInc, biasTop, optTop, batchSize, lr_scale, mom_scale);
		} // for (int epoch = 0; epoch < opt1.numEpochs; epoch++)

		hipDeviceSynchronize();
		cost /= CPUTrain.size();
		cost1 /= CPUTrain.size();
		printf("\nfinished epoch %d of %d; classify precision = %f; objective = %f; elapsed time = %f seconds\n", epoch, opt1.numEpochs,
				cost, cost1, (float)(clock() - startClock)/CLOCKS_PER_SEC);
		fprintf(pFile, "\nfinished epoch %d of %d; classify precision = %f; objective = %f; elapsed time = %f seconds\n", epoch, opt1.numEpochs,
				cost, cost1, (float)(clock() - startClock)/CLOCKS_PER_SEC);

		/*
		weight1.apply(NVMatrixOps::Abs(), absM);
		weightAbs1 = absM.sum() / absM.getNumElements();
		weight2.apply(NVMatrixOps::Abs(), absM);
		weightAbs2 = absM.sum() / absM.getNumElements();
		weight3.apply(NVMatrixOps::Abs(), absM);
		weightAbs3 = absM.sum() / absM.getNumElements();
		weight4.apply(NVMatrixOps::Abs(), absM);
		weightAbs4 = absM.sum() / absM.getNumElements();
		weightTop.apply(NVMatrixOps::Abs(), absM);
		weightAbsTop = absM.sum() / absM.getNumElements();


		weight1Inc.apply(NVMatrixOps::Abs(), absM);
		weightGradAbs1 = absM.sum() / absM.getNumElements();
		weight2Inc.apply(NVMatrixOps::Abs(), absM);
		weightGradAbs2 = absM.sum() / absM.getNumElements();
		weight3Inc.apply(NVMatrixOps::Abs(), absM);
		weightGradAbs3 = absM.sum() / absM.getNumElements();
		weight4Inc.apply(NVMatrixOps::Abs(), absM);
		weightGradAbs4 = absM.sum() / absM.getNumElements();
		weightTopInc.apply(NVMatrixOps::Abs(), absM);
		weightGradAbsTop = absM.sum() / absM.getNumElements();

		bias1.apply(NVMatrixOps::Abs(), absM);
		biasAbs1 = absM.sum() / absM.getNumElements();
		bias2.apply(NVMatrixOps::Abs(), absM);
		biasAbs2 = absM.sum() / absM.getNumElements();
		bias3.apply(NVMatrixOps::Abs(), absM);
		biasAbs3 = absM.sum() / absM.getNumElements();
		bias4.apply(NVMatrixOps::Abs(), absM);
		biasAbs4 = absM.sum() / absM.getNumElements();
		biasTop.apply(NVMatrixOps::Abs(), absM);
		biasAbsTop = absM.sum() / absM.getNumElements();

		bias1Inc.apply(NVMatrixOps::Abs(), absM);
		biasGradAbs1 = absM.sum() / absM.getNumElements();
		bias2Inc.apply(NVMatrixOps::Abs(), absM);
		biasGradAbs2 = absM.sum() / absM.getNumElements();
		bias3Inc.apply(NVMatrixOps::Abs(), absM);
		biasGradAbs3 = absM.sum() / absM.getNumElements();
		bias4Inc.apply(NVMatrixOps::Abs(), absM);
		biasGradAbs4 = absM.sum() / absM.getNumElements();
		biasTopInc.apply(NVMatrixOps::Abs(), absM);
		biasGradAbsTop = absM.sum() / absM.getNumElements();


		printf("weight abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", weightAbs1, weightAbs2, weightAbs3, weightAbs4, weightAbsTop);
		printf("weight grad abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", weightGradAbs1, weightGradAbs2, weightGradAbs3, weightGradAbs4, weightGradAbsTop);
		printf("bias abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", biasAbs1, biasAbs2, biasAbs3, biasAbs4, biasAbsTop);
		printf("bias grad abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", biasGradAbs1, biasGradAbs2, biasGradAbs3, biasGradAbs4, biasGradAbsTop);

		fprintf(pFile, "weight abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", weightAbs1, weightAbs2, weightAbs3, weightAbs4, weightAbsTop);
		fprintf(pFile, "weight grad abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", weightGradAbs1, weightGradAbs2, weightGradAbs3, weightGradAbs4, weightGradAbsTop);
		fprintf(pFile, "bias abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", biasAbs1, biasAbs2, biasAbs3, biasAbs4, biasAbsTop);
		fprintf(pFile, "bias grad abs: 1--%f, 2--%f, 3--%f, 4--%f, top--%f\n", biasGradAbs1, biasGradAbs2, biasGradAbs3, biasGradAbs4, biasGradAbsTop);
		 */

		// process the test set every 3 epochs
		if (epoch % 3 == 2) {
			hipDeviceSynchronize();
			startClock = clock();
			cost = 0;
			cost1 = 0;
			for (int batch = 0; batch < GPUTest.size(); batch++) {
				batchSize = GPUTest[batch]->getNumCols();
				// ====forward pass====
				// 0->1
				//cout << "0->1\n";
				//original
				activateConv(*GPUTest[batch], act1, weight1, bias1, opt1);
				act1.apply(ReluOperator());
				act1Pool.transpose(false);
				convLocalPool(act1, act1Pool, opt1.numFilters, opt1.poolSize, opt1.poolStartX, opt1.poolStride, opt1.poolOutX, MaxPooler());
				convResponseNormCrossMap(act1Pool, act1Denom, act1PoolNorm, opt1.numFilters, opt1.sizeF, opt1.addScale/opt1.sizeF, opt1.powScale, false);

				// 1->2
				//cout << "1->2\n";
				//original
				activateConv(act1PoolNorm, act2, weight2, bias2, opt2);
				act2.apply(ReluOperator());
				convResponseNormCrossMap(act2, act2Denom, act2Norm, opt2.numFilters, opt2.sizeF, opt2.addScale/opt2.sizeF, opt2.powScale, false);
				act2NormPool.transpose(false);
				convLocalPool(act2Norm, act2NormPool, opt2.numFilters, opt2.poolSize, opt2.poolStartX, opt2.poolStride, opt2.poolOutX, MaxPooler());

				// 2->3
				//cout << "2->3\n";
				// original
				activateLocal(act2NormPool, act3, weight3, bias3, opt3);
				act3.apply(ReluOperator());

				// 3->4
				//cout << "3->4\n";
				// original
				activateLocal(act3, act4, weight4, bias4, opt4);
				act4.apply(ReluOperator());

				// 4->top
				//cout << "4->top\n";
				actTop.transpose(true);
				actTop.resize(batchSize, opt1.labelSize);
				activate(act4, actTop, weightTop, biasTop, 0, 1);

				//softmax layer
				NVMatrix& max = actTop.max(1);
				actTop.addVector(max, -1);
				actTop.apply(NVMatrixOps::Exp());
				NVMatrix& sum = actTop.sum(1);
				actTop.eltwiseDivideByVector(sum);
				delete &max;
				delete &sum;

				// compute cost
				computeLogregSoftmaxGrad(*GPURawLabelTest[batch], actTop, actTopGrad, false, 1);
				actTop.transpose(false);
				computeLogregCost(*GPURawLabelTest[batch], actTop, trueLabelLogProbs, correctProbs); //labelLogProbs:(1, numCases); correctProbs:(1, numCases)
				cost += correctProbs.sum() / batchSize;
				cost1 += trueLabelLogProbs.sum() / batchSize;

			} //for (int batch = opt1.batchNum; batch < opt1.batchNum+opt1.testBatchNum; batch++)
			hipDeviceSynchronize();
			cost /= GPUTest.size();
			cost1 /= GPUTest.size();
			printf("\ntest set precision: %f\n; objective = %f; time elapsed = %f seconds\n", cost, cost1,
					(float)(clock() - startClock)/CLOCKS_PER_SEC);
			fprintf(pFile, "\ntest set precision: %f\n; objective = %f; time elapsed = %f seconds\n", cost, cost1,
					(float)(clock() - startClock)/CLOCKS_PER_SEC);

			// save checkpoint
			char* weight1File = "/scratch0/qwang37/cifar-10-batches-bin/weight1.bin", *bias1File = "/scratch0/qwang37/cifar-10-batches-bin/bias1.bin";
			char* weight2File = "/scratch0/qwang37/cifar-10-batches-bin/weight2.bin", *bias2File = "/scratch0/qwang37/cifar-10-batches-bin/bias2.bin";
			char* weight3File = "/scratch0/qwang37/cifar-10-batches-bin/weight3.bin", *bias3File = "/scratch0/qwang37/cifar-10-batches-bin/bias3.bin";
			char* weight4File = "/scratch0/qwang37/cifar-10-batches-bin/weight4.bin", *bias4File = "/scratch0/qwang37/cifar-10-batches-bin/bias4.bin";
			char* weightTopFile = "/scratch0/qwang37/cifar-10-batches-bin/weightTop.bin", *biasTopFile = "/scratch0/qwang37/cifar-10-batches-bin/biasTop.bin";

			NVSaveToFile(weight1, weight1File); NVSaveToFile(bias1, bias1File);
			NVSaveToFile(weight2, weight2File); NVSaveToFile(bias2, bias2File);
			NVSaveToFile(weight3, weight3File); NVSaveToFile(bias3, bias3File);
			NVSaveToFile(weight4, weight4File); NVSaveToFile(bias4, bias4File);
			NVSaveToFile(weightTop, weightTopFile); NVSaveToFile(biasTop, biasTopFile);
			printf("Checkpoint saved!\n\n");
			fprintf(pFile, "Checkpoint saved!\n\n");

		} //if (epoch % 10 == 0)

	} // for (int epoch = 0; epoch < opt1.numEpochs; epoch++)
	printf("finetuning_rnorm() complete!\n");
	fprintf(pFile, "finetuning_rnorm() complete!\n");
} // int finetune_rnorm()

void multiViewTest() {
	////assignOpt();
	printf("starting multiViewTest()!\n");
	fprintf(pFile, "starting multiViewTest()!\n");

	// initialize cublas
	hipSetDevice(cutGetMaxGflopsDeviceId());
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	cublasInit();

	// data and parameters storage
	NVMatrix act1, act1Pool, act1PoolNorm, act1Denom;
	NVMatrix act2, act2Norm, act2NormPool, act2Denom;
	NVMatrix act3;
	NVMatrix act4;
	NVMatrix actTop;
	NVMatrix softMaxAct;

	NVMatrix weight1, weight2, weight3, weight4, weightTop;
	NVMatrix bias1, bias2, bias3, bias4, biasTop; // bias4 is just an all-zero dummy vector
	// initialize parameters

	weight1.resize(opt1.numVis, opt1.numFilters);
	weight2.resize(opt2.numVis, opt2.numFilters);
	weight3.resize(opt3.numVis * opt3.outX * opt3.outX, opt3.numFilters);
	weight4.resize(opt4.numVis * opt4.outX * opt4.outX, opt4.numFilters);
	weightTop.resize(optTop.numVis, optTop.numFilters);

	bias1.resize(opt1.numFilters, 1);
	bias2.resize(opt2.numFilters, 1);
	bias3.resize(opt3.numFilters * opt3.outX * opt3.outX, 1);
	bias4.resize(opt4.numFilters * opt4.outX * opt4.outX, 1);
	biasTop.resize(1, optTop.numFilters);
	biasTop.setTrans(true);

	NVReadFromFile(weight1, "/scratch0/qwang37/cifar-10-batches-bin/weight1.bin");
	NVReadFromFile(weight2, "/scratch0/qwang37/cifar-10-batches-bin/weight2.bin");
	NVReadFromFile(weight3, "/scratch0/qwang37/cifar-10-batches-bin/weight3.bin");
	NVReadFromFile(weight4, "/scratch0/qwang37/cifar-10-batches-bin/weight4.bin");
	NVReadFromFile(weightTop, "/scratch0/qwang37/cifar-10-batches-bin/weightTop.bin");

	NVReadFromFile(bias1, "/scratch0/qwang37/cifar-10-batches-bin/bias1.bin");
	NVReadFromFile(bias2, "/scratch0/qwang37/cifar-10-batches-bin/bias2.bin");
	NVReadFromFile(bias3, "/scratch0/qwang37/cifar-10-batches-bin/bias3.bin");
	NVReadFromFile(bias4, "/scratch0/qwang37/cifar-10-batches-bin/bias4.bin");
	NVReadFromFile(biasTop, "/scratch0/qwang37/cifar-10-batches-bin/biasTop.bin");

	// read data to host memory (and labels to the GPU memory)
	int imPixels = 32*32*opt1.numChannels;
	int batchSize = opt1.batchSize;
	int testBatchNum = opt1.numTest / batchSize;
	vector<Matrix*> CPUTest(testBatchNum);
	vector<NVMatrix*> GPUTest(testBatchNum*opt1.numViews);
	vector<NVMatrix*> GPURawLabelTest(testBatchNum);

	// test set
	batchSize = opt1.batchSize;
	for (int batch = 0; batch < testBatchNum; batch++) {
		CPUTest[batch] = new Matrix(imPixels, batchSize);
		CPUTest[batch]->setTrans(false);
		for (int r = 0; r < opt1.numViews; r++)
			GPUTest[batch*opt1.numViews+r] = new NVMatrix();
		hmReadFromFile(*CPUTest[batch], opt1.dataPath + "/cifar_raw.bin", opt1.numTrain+batch*batchSize);
		GPURawLabelTest[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTest[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest[batch], opt1.dataPath + "/cifar_labels.bin", opt1.numTrain+batch*batchSize);
	}
	batchSize = opt1.numTest % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTest.push_back(new Matrix(imPixels, batchSize));
		CPUTest.back()->setTrans(false);
		for (int r = 0; r < opt1.numViews; r++)
			GPUTest.push_back(new NVMatrix());
		hmReadFromFile(*CPUTest.back(), opt1.dataPath + "/cifar_raw.bin", opt1.numTrain+testBatchNum*batchSize);
		GPURawLabelTest.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTest.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest.back(), opt1.dataPath + "/cifar_labels.bin", opt1.numTrain+testBatchNum*batchSize);
	}

	multiViewDataProvider(CPUTest, GPUTest, opt1, opt1.numViews, opt1.whitened); // copy data to the GPU side

	NVMatrix trueLabelLogProbs;
	NVMatrix correctProbs;
	MTYPE cost; // as before, we trace the performance using the cost variable
	MTYPE cost1;
	clock_t startClock;
	clock_t tick;
	cost = 0;
	cost1 = 0;
	hipDeviceSynchronize();
	startClock = clock();

	for (int batch = 0; batch < CPUTest.size(); batch++) {
		batchSize = CPUTest[batch]->getNumCols();
		for (int r = 0; r < opt1.numViews; r++) {
			// ====forward pass====
			// 0->1
			//cout << "0->1\n";
			//original
			activateConv(*GPUTest[batch*opt1.numViews+r], act1, weight1, bias1, opt1);
			act1.apply(ReluOperator());
			act1Pool.transpose(false);
			convLocalPool(act1, act1Pool, opt1.numFilters, opt1.poolSize, opt1.poolStartX, opt1.poolStride, opt1.poolOutX, MaxPooler());
			convResponseNormCrossMap(act1Pool, act1Denom, act1PoolNorm, opt1.numFilters, opt1.sizeF, opt1.addScale/opt1.sizeF, opt1.powScale, false);

			// 1->2
			//cout << "1->2\n";
			//original
			activateConv(act1PoolNorm, act2, weight2, bias2, opt2);
			act2.apply(ReluOperator());
			convResponseNormCrossMap(act2, act2Denom, act2Norm, opt2.numFilters, opt2.sizeF, opt2.addScale/opt2.sizeF, opt2.powScale, false);
			act2NormPool.transpose(false);
			convLocalPool(act2Norm, act2NormPool, opt2.numFilters, opt2.poolSize, opt2.poolStartX, opt2.poolStride, opt2.poolOutX, MaxPooler());

			// 2->3
			//cout << "2->3\n";
			// original
			activateLocal(act2NormPool, act3, weight3, bias3, opt3);
			act3.apply(ReluOperator());

			// 3->4
			//cout << "3->4\n";
			// original
			activateLocal(act3, act4, weight4, bias4, opt4);
			act4.apply(ReluOperator());

			// 4->top
			//cout << "4->top\n";
			actTop.transpose(true);
			actTop.resize(batchSize, opt1.labelSize);
			activate(act4, actTop, weightTop, biasTop, 0, 1);

			//softmax layer
			NVMatrix& max = actTop.max(1);
			actTop.addVector(max, -1);
			actTop.apply(NVMatrixOps::Exp());
			NVMatrix& sum = actTop.sum(1);
			actTop.eltwiseDivideByVector(sum);
			delete &max;
			delete &sum;
			actTop.transpose(false);

			if (r == 0)
				actTop.copy(softMaxAct);
			else
				softMaxAct.add(actTop);
		}// for (r = 0:9)
		softMaxAct.scale(0.1);
		computeLogregCost(*GPURawLabelTest[batch], softMaxAct, trueLabelLogProbs, correctProbs); //labelLogProbs:(1, numCases); correctProbs:(1, numCases)
		cost += correctProbs.sum();
		cost1 += trueLabelLogProbs.sum();
	}//for (batches)

	hipDeviceSynchronize();
	cost /= opt1.numTest;
	cost1 /= opt1.numTest;
	printf("\ntest set precision: %f\n; objective = %f; time elapsed = %f seconds\n", cost, cost1,
			(float)(clock() - startClock)/CLOCKS_PER_SEC);
	printf("multiViewTest() complete!\n");

	fprintf(pFile, "\ntest set precision: %f\n; objective = %f; time elapsed = %f seconds\n", cost, cost1,
			(float)(clock() - startClock)/CLOCKS_PER_SEC);
	fprintf(pFile, "multiViewTest() complete!\n");
} // void multiViewTest()


void testCropDataProvider() {
	//assignOpt();
	printf("starting test cropDataProvider()!\n");
	// read data to host memory (and labels to the GPU memory)
	int imPixels = 32*32*opt1.numChannels;
	int batchSize = opt1.batchSize;
	int trainBatchNum = opt1.numTrain / batchSize;
	int testBatchNum = opt1.numTest / batchSize;
	vector<Matrix*> CPUTrain(trainBatchNum), CPUTest(testBatchNum);
	vector<NVMatrix*> GPUTrain(trainBatchNum), GPUTest(testBatchNum);
	vector<NVMatrix*> GPURawLabelTrain(trainBatchNum), GPURawLabelTest(testBatchNum);

	for (int batch = 0; batch < trainBatchNum; batch++) {
		CPUTrain[batch] = new Matrix(imPixels, batchSize);
		CPUTrain[batch]->setTrans(false);
		GPUTrain[batch] = new NVMatrix();
		hmReadFromFile(*CPUTrain[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", batch*batchSize);
		GPURawLabelTrain[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTrain[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTrain[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", batch*batchSize);
	}
	batchSize = opt1.numTrain % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTrain.push_back(new Matrix(imPixels, batchSize));
		CPUTrain.back()->setTrans(false);
		GPUTrain.push_back(new NVMatrix());
		hmReadFromFile(*CPUTrain.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", trainBatchNum*batchSize);
		GPURawLabelTrain.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTrain.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTrain.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", trainBatchNum*batchSize);
	}
	// test set
	batchSize = opt1.batchSize;
	for (int batch = 0; batch < testBatchNum; batch++) {
		CPUTest[batch] = new Matrix(imPixels, batchSize);
		CPUTest[batch]->setTrans(false);
		GPUTest[batch] = new NVMatrix();
		hmReadFromFile(*CPUTest[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", opt1.numTrain+batch*batchSize);
		GPURawLabelTest[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTest[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", opt1.numTrain+batch*batchSize);
	}
	batchSize = opt1.numTest % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTest.push_back(new Matrix(imPixels, batchSize));
		CPUTest.back()->setTrans(false);
		GPUTest.push_back(new NVMatrix());
		hmReadFromFile(*CPUTest.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", opt1.numTrain+testBatchNum*batchSize);
		GPURawLabelTest.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTest.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", opt1.numTrain+testBatchNum*batchSize);
	}

	cropDataProvider(CPUTrain, GPUTrain, opt1, false, false);
	remove("/scratch0/qwang37/cifar-10-batches-bin/test_out.bin");
	for (int batch = 0; batch < GPUTrain.size(); batch++) {
		NVSaveToFile(*GPUTrain[batch], "/scratch0/qwang37/cifar-10-batches-bin/test_out.bin", true);
	}
	printf("cropDataProvider() test done!\n");
}

void testNVReadFromFileUint8() {
	//assignOpt();
	printf("starting test testNVReadFromFileUint8()!\n");
	// read data to host memory (and labels to the GPU memory)
	int imPixels = 32*32*opt1.numChannels;
	int batchSize = opt1.batchSize;
	int trainBatchNum = opt1.numTrain / batchSize;
	int testBatchNum = opt1.numTest / batchSize;
	vector<Matrix*> CPUTrain(trainBatchNum), CPUTest(testBatchNum);
	vector<NVMatrix*> GPUTrain(trainBatchNum), GPUTest(testBatchNum);
	vector<NVMatrix*> GPURawLabelTrain(trainBatchNum), GPURawLabelTest(testBatchNum);

	for (int batch = 0; batch < trainBatchNum; batch++) {
		CPUTrain[batch] = new Matrix(imPixels, batchSize);
		CPUTrain[batch]->setTrans(false);
		GPUTrain[batch] = new NVMatrix();
		hmReadFromFile(*CPUTrain[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", batch*batchSize);
		GPURawLabelTrain[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTrain[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTrain[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", batch*batchSize);
	}
	batchSize = opt1.numTrain % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTrain.push_back(new Matrix(imPixels, batchSize));
		CPUTrain.back()->setTrans(false);
		GPUTrain.push_back(new NVMatrix());
		hmReadFromFile(*CPUTrain.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", trainBatchNum*batchSize);
		GPURawLabelTrain.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTrain.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTrain.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", trainBatchNum*batchSize);
	}
	// test set
	batchSize = opt1.batchSize;
	for (int batch = 0; batch < testBatchNum; batch++) {
		CPUTest[batch] = new Matrix(imPixels, batchSize);
		CPUTest[batch]->setTrans(false);
		GPUTest[batch] = new NVMatrix();
		hmReadFromFile(*CPUTest[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", opt1.numTrain+batch*batchSize);
		GPURawLabelTest[batch] = new NVMatrix(1, batchSize);
		GPURawLabelTest[batch]->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest[batch], "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", opt1.numTrain+batch*batchSize);
	}
	batchSize = opt1.numTest % opt1.batchSize; // the last batch
	if (batchSize > 0) {
		CPUTest.push_back(new Matrix(imPixels, batchSize));
		CPUTest.back()->setTrans(false);
		GPUTest.push_back(new NVMatrix());
		hmReadFromFile(*CPUTest.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", opt1.numTrain+testBatchNum*batchSize);
		GPURawLabelTest.push_back(new NVMatrix(1, batchSize));
		GPURawLabelTest.back()->setTrans(false);
		NVRawLabelReadFromFile(*GPURawLabelTest.back(), "/scratch0/qwang37/cifar-10-batches-bin/cifar_labels.bin", opt1.numTrain+testBatchNum*batchSize);
	}
	cropDataProvider(CPUTrain, GPUTrain, opt1, false, false);
	remove("/scratch0/qwang37/cifar-10-batches-bin/test_out.bin");
	for (int batch = 0; batch < GPUTrain.size(); batch++) {
		NVSaveToFile(*GPUTrain[batch], "/scratch0/qwang37/cifar-10-batches-bin/test_out.bin", true);
	}
	printf("testNVReadFromFileUint8() test done!\n");
}

void centerData() {
	ifstream in;
	printf("starting centering data\n");
	MTYPE* data = (MTYPE*) malloc(60000*3072*sizeof(MTYPE));
	MTYPE* mean = (MTYPE*) malloc(3072*sizeof(MTYPE));
	char* labels = (char*) malloc(60000*sizeof(char));
	char dir_name[] = "/scratch0/qwang37/cifar-10-batches-bin/";
	char file_name[] = "data_batch_1.bin";
	char full_name[100];
	int record_start;

	for (int j = 0; j < 3072; j++)
		mean[j] = 0.0;

	for (int k = 1; k <= 5; k++) {
		file_name[11] = '0' + k;
		strcpy(full_name, dir_name);
		strcat(full_name, file_name);
		in.open(full_name, std::ifstream::in | std::ifstream::binary);
		if (in.fail()) {
			printf("open data file %d failed!\n", k);
			exit(-1);
		}
		printf("reading batch %d\n", k);

		for (int i = 0; i < 10000; i++) {
			record_start = (k-1)*10000 + i;
			labels[record_start] = in.get();
			for (int j = 0; j < 3072; j++) {
				data[record_start*3072+j] = MTYPE(in.get());
				mean[j] += data[record_start*3072+j];
			}
		}
		in.close();
	}



	char test_name[100];
	strcpy(test_name, dir_name);
	strcat(test_name, "test_batch.bin");
	in.open(test_name, std::ifstream::in | std::ifstream::binary);
	printf("reading test batch\n");

	for (int i = 0; i < 10000; i++) {
		record_start = 5*10000 + i;
		labels[record_start] = in.get();
		for (int j = 0; j < 3072; j++) {
			data[record_start*3072+j] = (MTYPE)in.get();
			//mean[j] += data[record_start*3072+j];
		}
	}
	in.close();

	/*
	for (int j = 0; j < 3072; j++)
		mean[j] /= 50000.0;
	*/
	ifstream in_mean;
	in_mean.open("/scratch0/qwang37/cifar-10-batches-bin/data_mean.bin", std::ifstream::in | std::ifstream::binary);
	if (in_mean.fail()) {
		cout << "open file failed!\n";
		return;
	}
	for (int j = 0; j < 3072; j++)
		in_mean.read((char*)(mean+j), 4);
	in_mean.close();

	for (int i = 0; i < 60000; i++)
		for (int j = 0; j < 3072; j++)
			data[i*3072+j] -= mean[j];
			//data[i*3072+j] -= 127.5;


	ofstream out("/scratch0/qwang37/cifar-10-batches-bin/cifar_centered.bin", std::ofstream::out | std::ifstream::binary);
	out.write((char*)data, 60000*3072*sizeof(MTYPE));
	out.close();
}

void convertToMTYPE() {
	ifstream in;
	printf("starting copying data\n");
	MTYPE* data = (MTYPE*) malloc(60000*3072*sizeof(MTYPE));
	char* labels = (char*) malloc(60000*sizeof(char));
	char dir_name[] = "/scratch0/qwang37/cifar-10-batches-bin/";
	char file_name[] = "data_batch_1.bin";
	char full_name[100];
	int record_start;

	printf("starting copy data\n");
	for (int k = 1; k <= 5; k++) {
		file_name[11] = '0' + k;
		strcpy(full_name, dir_name);
		strcat(full_name, file_name);
		in.open(full_name, std::ifstream::in | std::ifstream::binary);
		if (in.fail()) {
			printf("open data file %d failed!\n", k);
			exit(-1);
		}
		printf("reading batch %d\n", k);

		for (int i = 0; i < 10000; i++) {
			record_start = (k-1)*10000 + i;
			labels[record_start] = in.get();
			for (int j = 0; j < 3072; j++) {
				data[record_start*3072+j] = MTYPE(in.get());
			}
		}
		in.close();
	}

	char test_name[100];
	strcpy(test_name, dir_name);
	strcat(test_name, "test_batch.bin");
	in.open(test_name);
	printf("reading test batch\n");

	for (int i = 0; i < 10000; i++) {
		record_start = 5*10000 + i;
		labels[record_start] = in.get();
		for (int j = 0; j < 3072; j++) {
			data[record_start*3072+j] = (MTYPE)in.get();
		}
	}
	in.close();

	ofstream out("/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin", std::ofstream::out | std::ifstream::binary);
	out.write((char*)data, 60000*3072*sizeof(MTYPE));
	out.close();
}

void testAssembleMatrix() {
	printf("start testAssembleMatrix()\n");
	vector<NVMatrix> matrices(4);
	Matrix tmp(2,3);
	MTYPE* data = tmp.getData();
	for (int i = 0; i < 4; i++)
		matrices[i] = new NVMatrix(2,3);
	// test1
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 6; j++)
			data[j] = i * 6 + j;
		tmp.setTrans(false);
		matrices[i].copyFromHost(tmp, true);
	}
	NVMatrix rowFalse;
	assembleNVMatrix(matrices, rowFalse, 0);
	rowFalse.printShape("rowFalse");
	rowFalse.print(8,3);

	for (int i = 0; i < 4; i++) {
		matrices[i].resize(0,0);
	}
	splitNVMatrix(matrices, rowFalse, 0);
	char a[10];
	for (int i = 0; i < 4; i++) {
		sprintf(a, "rowFalse%d", i);
		matrices[i].printShape(a);
		matrices[i].print(2,3);
	}

	// test2
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 6; j++)
			data[j] = i * 6 + j;
		tmp.setTrans(true);
		matrices[i].copyFromHost(tmp, true);
	}
	NVMatrix rowTrue;
	assembleNVMatrix(matrices, rowTrue, 0);
	rowTrue.printShape("rowTrue");
	rowTrue.print(8,3);

	for (int i = 0; i < 4; i++) {
		matrices[i].resize(0,0);
	}
	splitNVMatrix(matrices, rowTrue, 0);
	for (int i = 0; i < 4; i++) {
		sprintf(a, "rowTrue%d", i);
		matrices[i].printShape(a);
		matrices[i].print(2,3);
	}

	// test3
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 6; j++)
			data[j] = i * 6 + j;
		tmp.setTrans(false);
		matrices[i].copyFromHost(tmp, true);
	}
	NVMatrix colFalse;
	assembleNVMatrix(matrices, colFalse, 1);
	colFalse.printShape("colFalse");
	colFalse.print(2,12);

	for (int i = 0; i < 4; i++) {
		matrices[i].resize(0,0);
	}
	splitNVMatrix(matrices, colFalse, 1);
	for (int i = 0; i < 4; i++) {
		sprintf(a, "colFalse%d", i);
		matrices[i].printShape(a);
		matrices[i].print(2,3);
	}

	// test4
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 6; j++)
			data[j] = i * 6 + j;
		tmp.setTrans(true);
		matrices[i].copyFromHost(tmp, true);
	}
	NVMatrix colTrue;
	assembleNVMatrix(matrices, colTrue, 1);
	colTrue.printShape("colTrue");
	colTrue.print(2,12);

	for (int i = 0; i < 4; i++) {
		matrices[i].resize(0,0);
	}
	splitNVMatrix(matrices, colTrue, 1);
	for (int i = 0; i < 4; i++) {
		sprintf(a, "colTrue%d", i);
		matrices[i].printShape(a);
		matrices[i].print(2,3);
	}

	printf("finished testAssembleMatrix()\n");
}

void testAssembleMatrix1() {
	printf("start testAssembleMatrix1()\n");
	vector<NVMatrix> matrices(4);
	NVMatrix mat(32*32*3, 60000);
	mat.setTrans(false);
	NVReadFromFile(mat, "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin");
	splitNVMatrix(matrices, mat, 0);
	for (int i = 0; i < 4; i++) {
		char a[100];
		sprintf(a, "/scratch0/qwang37/cifar-10-batches-bin/testAssemble/r%d.bin", i);
		NVSaveToFile(matrices[i], a);
	}
	NVMatrix ass;
	assembleNVMatrix(matrices, ass, 0);
	NVSaveToFile(ass, "/scratch0/qwang37/cifar-10-batches-bin/testAssemble.bin");
	printf("completed testAssembleMatrix1()!\n");
}

void testAssembleMatrix2() {
	printf("start testAssembleMatrix2()\n");
	vector<NVMatrix> sub(2);
	NVMatrix mat(32*32*3, 60000);
	mat.setTrans(false);
	NVReadFromFile(mat, "/scratch0/qwang37/cifar-10-batches-bin/cifar_raw.bin");
	splitNVMatrix(sub[0], sub[1], mat, 1000, 2072, 0);
	for (int i = 0; i < 2; i++) {
		char a[100];
		sprintf(a, "/scratch0/qwang37/cifar-10-batches-bin/testAssemble/r%d.bin", i);
		NVSaveToFile(sub[i], a);
	}
	NVMatrix ass;
	assembleNVMatrix(sub[0], sub[1], ass, 0);
	printf("finished assembleNVMatrix!\n");
	NVSaveToFile(ass, "/scratch0/qwang37/cifar-10-batches-bin/rowAssemble.bin");

	splitNVMatrix(sub[0], sub[1], mat, 20000, 40000, 1);
	for (int i = 0; i < 2; i++) {
		char a[100];
		sprintf(a, "/scratch0/qwang37/cifar-10-batches-bin/testAssemble/c%d.bin", i);
		NVSaveToFile(sub[i], a);
	}
	NVMatrix ass1;
	assembleNVMatrix(sub[0], sub[1], ass1, 1);
	NVSaveToFile(ass1, "/scratch0/qwang37/cifar-10-batches-bin/colAssemble.bin");
	printf("completed testAssembleMatrix2()!\n");
}

void testGenFilterMask() {
	hiprandState* devStates = init_cuda_rand(100);
	printf("start testGenFilterMask()!\n");
	NVMatrix a;
	genFilterMask(a, 10, 10, 0.2, devStates);
	a.printShape("a");
	a.print(10,10);
}

void testAbs() {
	NVMatrix a;
	a.resize(16*10,10);
	a.setTrans(false);
	a.randomizeGaussian();
	NVMatrix b;
	NVMask maxMask;
	convCompeteAbs(a, b, maxMask, 10, 2, 2, 4, 10);
	b.print(16*10, 10);
}
